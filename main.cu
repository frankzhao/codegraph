/* CODEGRAPH GENERATED CODE BEGIN */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void codegraphKernel(float* a, float* c, const int chunkSize, const int limit) {
    int threadid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    // Don't calculate for elements outside of matrix
    if (threadid >= chunkSize)
    	return;

    int chunkidx = threadid * chunkSize;
    
    // Calculate
    c[threadid] = a[chunkidx + 0]  +  a[chunkidx + 1]  *  a[chunkidx + 2]  +  a[chunkidx + 3]  *  a[chunkidx + 4];
}
int main() {
    const int chunkSize = 5;
    const int initSize = 10;
    const int limit = (int) initSize/chunkSize;
    float initmem[10] = {
        (float) 2, (float) 1, (float) 0, (float) 2, (float) 3, (float) 0, (float) 2, (float) 3, (float) 4, (float) 3
    };


    // Copy to device
	  float* dev_initmem = 0;
	  float* dev_out = 0;
    float out[2];
    hipMalloc(&dev_initmem, initSize * sizeof(float));
    hipMalloc(&dev_out, 2 * sizeof(float));

    hipMemcpy(dev_initmem, initmem, initSize * sizeof(float), hipMemcpyHostToDevice);

    // Run on device
    codegraphKernel<<<1,initSize>>>(dev_initmem, dev_out, chunkSize, limit);

    // Copy results
    hipMemcpy(out, dev_out, 2 * sizeof(float), hipMemcpyDeviceToHost);

    /*
     *Do something with results here
     */

    // Free
 	  hipFree(dev_initmem);
 	  hipFree(dev_out);
}
/* CODEGRAPH GENERATED CODE END */
