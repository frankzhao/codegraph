/* CODEGRAPH GENERATED CODE BEGIN */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

__global__ void codegraphKernel(float* a, float* c, const int chunkSize, const int limit) {
    int threadid = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    // Don't calculate for elements outside of matrix
    if (threadid >= limit)
    	return;

    int chunkidx = threadid * chunkSize;
    
    // Calculate
    c[threadid] = a[chunkidx + 0]  +  a[chunkidx + 1]  *  a[chunkidx + 2]  +  a[chunkidx + 3]  *  a[chunkidx + 4];
}
int main() {
    const int chunkSize = 5;
    const int initSize = 20;
    const int limit = (int) initSize/chunkSize;
    float initmem[20] = {
        (float) 0, (float) 1, (float) 2, (float) 3, (float) 2, (float) 0, (float) 3, (float) 1, (float) 2, (float) 2, (float) 0, (float) 2, (float) 3, (float) 3, (float) 4, (float) 0, (float) 3, (float) 3, (float) 4, (float) 2
    };


    // Copy to device
	  float* dev_initmem = 0;
	  float* dev_out = 0;
    float out[4];
    hipMalloc(&dev_initmem, initSize * sizeof(float));
    hipMalloc(&dev_out, 4 * sizeof(float));

    hipMemcpy(dev_initmem, initmem, initSize * sizeof(float), hipMemcpyHostToDevice);

    // Run on device
    codegraphKernel<<<1,initSize>>>(dev_initmem, dev_out, chunkSize, limit);

    // Copy results
    hipMemcpy(out, dev_out, 4 * sizeof(float), hipMemcpyDeviceToHost);

    /*
     *Do something with results here
     */
    printf("%f %f %f %f", out[0], out[1], out[2], out[3]);

    // Free
 	  hipFree(dev_initmem);
 	  hipFree(dev_out);
}
/* CODEGRAPH GENERATED CODE END */
